#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuDabgdxyz_krn(int n, double alpha, double beta, double gamma, double delta,
				  double* x, double *y, double *z)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	double t;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  if (beta == 0.0)
	    t = PREC_DMUL(alpha,x[id]);
	  else
	    t = PREC_DADD(PREC_DMUL(alpha, x[id]), PREC_DMUL(beta,y[id]));
	  if (delta == 0.0)
	    z[id] = gamma * t;
	  else
	    z[id] = PREC_DADD(PREC_DMUL(gamma, t), PREC_DMUL(delta,z[id]));
	  y[id] = t;
	}
}


void spgpuDabgdxyz(spgpuHandle_t handle,
		   int n,
		   double alpha,
		   double beta,
		   double gamma,
		   double delta,
		   __device double* x,
		   __device double* y,
		   __device double *z)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuDabgdxyz_krn<<<grid, block, 0, handle->currentStream>>>(n, alpha, beta, gamma, delta,
								   x, y, z);
}

