/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2015
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include "hip/hip_complex.h"

extern "C"
{
#include "core.h"
#include "dia.h"
}

#include "debug.h"

#define VALUE_TYPE hipFloatComplex
#define TYPE_SYMBOL C
#define TEX_FETCH_TYPE hipFloatComplex
#include "dia_spmv_base.cuh"

