#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuCxyzw_krn(int n, hipFloatComplex  a, hipFloatComplex  b,
			       hipFloatComplex  c, hipFloatComplex  d,
			       hipFloatComplex  e, hipFloatComplex  f,
			       hipFloatComplex * x, hipFloatComplex  *y,
			       hipFloatComplex  *z, hipFloatComplex  *w)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	hipFloatComplex  ty, tz;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  ty    = hipCfmaf(a, x[id], hipCmulf(b,y[id]));
	  tz    = hipCfmaf(c, ty, hipCmulf(d,z[id]));
	  w[id] = hipCfmaf(e, tz, hipCmulf(f,w[id]));
	  y[id] = ty;
	  z[id] = tz;
	}
}


void spgpuCxyzw(spgpuHandle_t handle,
		int n,
		hipFloatComplex  a, hipFloatComplex  b,
		hipFloatComplex  c, hipFloatComplex  d,
		hipFloatComplex  e, hipFloatComplex  f,
		__device hipFloatComplex * x,
		__device hipFloatComplex * y,
		__device hipFloatComplex * z,
		__device hipFloatComplex *w)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuCxyzw_krn<<<grid, block, 0, handle->currentStream>>>(n, a,b,c,d,e,f,
								  x, y, z,w);
}

