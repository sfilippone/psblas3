#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"
#include "hip/hip_complex.h"


extern "C"
{
#include "core.h"
#include "vector.h"
}


//#define USE_CUBLAS

//#define ASSUME_LOCK_SYNC_PARALLELISM


#define BLOCK_SIZE 512

static __device__ double dnrm2ReductionResult[128];

__global__ void spgpuZnrm2_kern(int n, hipDoubleComplex* x)
{
	__shared__ double sSum[BLOCK_SIZE];

	double res = 0;

	hipDoubleComplex* lastX = x + n;

	x += threadIdx.x + blockIdx.x*BLOCK_SIZE;

	int blockOffset = gridDim.x*BLOCK_SIZE;

	while (x < lastX)
    	{
		hipDoubleComplex x1 = x[0];
		res = res + hipCreal(hipCmul(x1,hipConj(x1)));
		
		x += blockOffset;

	}

	if (threadIdx.x >= 32)
		sSum[threadIdx.x] = res;

	__syncthreads();


	// Start reduction!

	if (threadIdx.x < 32) 
	{
		for (int i=1; i<BLOCK_SIZE/32; ++i)
		{
			res += sSum[i*32 + threadIdx.x];
		}

	//useless (because inter-warp)
#ifndef	ASSUME_LOCK_SYNC_PARALLELISM
	}
	__syncthreads(); 

	if (threadIdx.x < 32) 
	{
#endif	

#ifdef ASSUME_LOCK_SYNC_PARALLELISM
		volatile double* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] += vsSum[threadIdx.x + 16];
		if (threadIdx.x < 8) vsSum[threadIdx.x] += vsSum[threadIdx.x + 8];
		if (threadIdx.x < 4) vsSum[threadIdx.x] += vsSum[threadIdx.x + 4];
		if (threadIdx.x < 2) vsSum[threadIdx.x] += vsSum[threadIdx.x + 2];
		if (threadIdx.x == 0)
			dnrm2ReductionResult[blockIdx.x] = vsSum[0] + vsSum[1];

#else
		double* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] += vsSum[threadIdx.x + 16];
		__syncthreads();
		if (threadIdx.x < 8) vsSum[threadIdx.x] += vsSum[threadIdx.x + 8];
		__syncthreads();
		if (threadIdx.x < 4) vsSum[threadIdx.x] += vsSum[threadIdx.x + 4];
		__syncthreads();
		if (threadIdx.x < 2) vsSum[threadIdx.x] += vsSum[threadIdx.x + 2];
		__syncthreads();
		if (threadIdx.x == 0)
			dnrm2ReductionResult[blockIdx.x] = vsSum[0] + vsSum[1];
#endif	
	}
}

double spgpuZnrm2(spgpuHandle_t handle, int n, hipDoubleComplex* x)
{
#ifdef USE_CUBLAS
	double res;
	hipblasDnrm2(n,x,1,&res);
	hipDeviceSynchronize();
	
	return res;

#else
	double res = 0;

#if 0 	
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device);	

	int blocks = min(128, min(prop.multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#else
	int blocks = min(128, min(handle->multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#endif
	
	double tRes[128];

	spgpuZnrm2_kern<<<blocks, BLOCK_SIZE, 0, handle->currentStream>>>(n, x);;
	hipMemcpyFromSymbol(tRes, HIP_SYMBOL(dnrm2ReductionResult),blocks*sizeof(double));

	for (int i=0; i<blocks; ++i)
	{
		res += tRes[i];
	}

	cudaCheckError("CUDA error on dnrm2");
	
	return sqrt(res);
#endif
}

void spgpuZmnrm2(spgpuHandle_t handle, double *y, int n, __device hipDoubleComplex *x, int count, int pitch)
{
	for (int i=0; i < count; ++i)
	{
		y[i] = spgpuZnrm2(handle, n, x);
		x += pitch;
	}
}
