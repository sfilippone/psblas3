#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include "hip/hip_complex.h"


extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512

#if 1
__global__ void spgpuZaxpby_krn(hipDoubleComplex *z, int n, hipDoubleComplex beta, hipDoubleComplex *y, hipDoubleComplex alpha, hipDoubleComplex* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (cuDoubleComplex_isZero(beta)) {
	  for ( ; id < n; id +=gridSize)
	    //if (id,n) 
	    {
	      // Since z, x and y are accessed with the same offset by the same thread,
	      // and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      
	      z[id] = hipCmul(alpha,x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    //if (id,n) 
	    {
	      z[id] = hipCfma(beta, y[id], hipCmul(alpha, x[id]));
	    }
	}
}

void spgpuZaxpby(spgpuHandle_t handle,
	__device hipDoubleComplex *z,
	int n,
	hipDoubleComplex beta,
	__device hipDoubleComplex *y,
	hipDoubleComplex alpha,
	__device hipDoubleComplex* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuZaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}
#else
__global__ void spgpuZaxpby_krn(hipDoubleComplex *z, int n, hipDoubleComplex beta, hipDoubleComplex *y, hipDoubleComplex alpha, hipDoubleComplex* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (cuDoubleComplex_isZero(beta))
			z[id] = hipCmul(alpha,x[id]);
		else
			z[id] = hipCfma(alpha, x[id], hipCmul(beta,y[id]));
	}
}


void spgpuZaxpby_(spgpuHandle_t handle,
	__device hipDoubleComplex *z,
	int n,
	hipDoubleComplex beta,
	__device hipDoubleComplex *y,
	hipDoubleComplex alpha,
	__device hipDoubleComplex* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuZaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuZaxpby(spgpuHandle_t handle,
	__device hipDoubleComplex *z,
	int n,
	hipDoubleComplex beta,
	__device hipDoubleComplex *y,
	hipDoubleComplex alpha,
	__device hipDoubleComplex* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpuZaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuZaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on daxpby");
}
#endif
void spgpuZmaxpby(spgpuHandle_t handle,
		  __device hipDoubleComplex *z,
		  int n,
		  hipDoubleComplex beta,
		  __device hipDoubleComplex *y,
		  hipDoubleComplex alpha,
		  __device hipDoubleComplex* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuZaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
