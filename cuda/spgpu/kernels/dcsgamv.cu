#include "hip/hip_runtime.h"
#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"
extern "C"
{
#include "core.h"
#include "csga.h"
}

#include "debug.h"

//#define MAX_NNZ_PER_WG 6144
#define MAX_NNZ_PER_WG 4096
#define THREAD_BLOCK   1024
#define MAX_GRID_SIZE 65536
#define WARP_SIZE 32


__device__ double warp_reduce(double val){
  for(int offset=warpSize/2; offset>0; offset/=2){
    val += __shfl_down_sync(0xffffffff,val, offset);
  }
  return val;
}

__global__ void dCSGAmvINNER(double* as, int* ja, int* irp, double* multivector,
			     int m, int n, int col_multivector,
			     int* rowBlocks, double* resultData, int baseIndex){
  __shared__ double vals[MAX_NNZ_PER_WG];
  __shared__ int cols[MAX_NNZ_PER_WG];
  
  int startRow = rowBlocks[blockIdx.x];
  int stopRow = rowBlocks[blockIdx.x+1];
  long int numRows = stopRow - startRow;
  int nnz = irp[stopRow]-irp[startRow];
  int tid = threadIdx.x; // indice del thread nel blocco
  if (numRows > 1){
    //CSR-Stream
    //printf("csr stream\n");
    
    int localCol;
    
    for (int i = tid; i < nnz; i+= blockDim.x){ 
      localCol = irp[startRow]+i;
      vals[i] = as[localCol];
      //vals[i] *= multivector[ja[localCol]*col_multivector+j];
      cols[i] = ja[localCol];
    }
    int firstCol = irp[startRow];
    
    __syncthreads();
    for (int t = tid; t < numRows*col_multivector; t += blockDim.x){
      int localRow = startRow + t/col_multivector;
      int j = t%col_multivector;
      double temp = 0; 
      for (int i = irp[localRow]-firstCol; i < irp[localRow+1]-firstCol; i++){
	temp += vals[i]*multivector[cols[i]*col_multivector + j];
      }
      resultData[localRow*col_multivector +j] = temp;
    }
    
    __syncthreads();    
    
  } else {
    //CSR-Vector
    //printf("csr vector\n");
    int warpId = tid / 32; // Global warp index
    int lane = tid &(32-1); // thread index within the warp
    //one warp per row
    double val; 
    int col;
    double sum[64] = {0};   
    if (nnz < 4096){
      int localCol;
      for (int i = tid; i < nnz; i+= blockDim.x){ 
	localCol = irp[startRow]+i;
	vals[i] = as[localCol];
	cols[i] = ja[localCol];
      }
    }
    __syncthreads();
    if (warpId < col_multivector){
      for (int col_m = warpId; col_m < col_multivector; col_m +=32){
	for (int i = irp[startRow] + lane; i < irp[startRow+1]; i +=32){
	  if (nnz < 4096){
	    val = vals[i-irp[startRow]];
	    col = cols[i-irp[startRow]];
	  } else {
	    val = as[i];
	    col = ja[i];
	  }
	  sum[col_m] += val*multivector[col*col_multivector + col_m];     
	}
	sum[col_m] = warp_reduce(sum[col_m]);
	if (lane == 0){
	  resultData[startRow*col_multivector + col_m] = sum[col_m];   
	}
      }
    }
  }
}


__host__ int dCSGAMV(spgpuHandle_t handle, 
	    double beta,
	    double* y, 
	    double alpha, 
	    const double* as, 
	    const int* ja,
	    const int* irp,
	    int m,
	    int n,
	    int  numBlocks,
	    const int* rowBlocks,
	    const double *x,
	    int baseIndex)
{
  int maxBForACall = max(handle->maxGridSizeX, numBlocks);
  int blockX = THREAD_BLOCK;
  int gridX = maxBForACall;
  int rp,rows, blcks, bp, numb;
  dim3 blockSize(blockX);
  dim3 gridSize(gridX);

  fprintf(stderr," dcsgamv  %d  %d \n",numBlocks,rowBlocks[0],rowBlocks[1]);

  bp = 0;
  rp = 0;
  numb = numBlocks;  
  while (numb > maxBForACall) {//managing large vectors
    blcks = maxBForACall;
    rp = rowBlocks[bp];
    rows = rowBlocks[bp+blcks]-rp;
    fprintf(stderr,"  rp %d  rows %d  bp %d  \n",rp,rows,bp);
    bp   += blcks;
    numb -= blcks;
  }
  blcks = numb;
  rp = rowBlocks[bp];
  rows = rowBlocks[bp+blcks]-rp;
  fprintf(stderr,"  rp %d  rows %d  bp %d  \n",rp,rows,bp);
  rp += rows;
  fprintf(stderr,"  Final  rows %d    \n",rows);
  return(SPGPU_SUCCESS);
}
