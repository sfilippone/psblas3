#include "hip/hip_runtime.h"
#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"
extern "C"
{
#include "core.h"
#include "csga.h"
}

#include "debug.h"

//#define MAX_NNZ_PER_WG 6144
#define MAX_NNZ_PER_WG 4096
#define THREAD_BLOCK   1024
#define MAX_GRID_SIZE 65536
#define WARP_SIZE 32


__device__ double warp_reduce(double val){
  for(int offset=warpSize/2; offset>0; offset/=2){
    val += __shfl_down_sync(0xffffffff,val, offset);
  }
  return val;
}

__global__ void dCSGAmvINNER(double alpha, const double* as, const int* ja, const int* irp,
			     const double* multivector, int m, int n, int col_multivector,
			     const int* rowBlocks, double beta, double* resultData, int baseIndex){
  __shared__ double vals[MAX_NNZ_PER_WG];
  __shared__ int cols[MAX_NNZ_PER_WG];
  
  int startRow = rowBlocks[blockIdx.x];
  int stopRow = rowBlocks[blockIdx.x+1];
  long int numRows = stopRow - startRow;
  int nnz = irp[stopRow]-irp[startRow];
  int tid = threadIdx.x; // indice del thread nel blocco

  if (numRows > 1){
    //CSR-Stream
    //printf("csr stream\n");
    
    int localCol;
    
    for (int i = tid; i < nnz; i+= blockDim.x){ 
      localCol = irp[startRow]+i;
      vals[i] = as[localCol];
      //vals[i] *= multivector[ja[localCol]*col_multivector+j];
      cols[i] = ja[localCol];
    }
    //return;
    int firstCol = irp[startRow];
    
    __syncthreads();
    for (int t = tid; t < numRows*col_multivector; t += blockDim.x){
      int localRow = startRow + t/col_multivector;
      int j = t%col_multivector;
      double temp = 0; 
      for (int i = irp[localRow]-firstCol; i < irp[localRow+1]-firstCol; i++){
	temp += vals[i]*multivector[cols[i]*col_multivector + j];
      }
      if (beta == 0.0) {
	resultData[localRow*col_multivector +j] = alpha*temp;
      } else {
	resultData[localRow*col_multivector +j] = alpha*temp + 	beta*resultData[localRow*col_multivector +j];
      }
    }
    
    __syncthreads();    
    
  } else {
    //CSR-Vector
    //printf("csr vector\n");
    int warpId = tid / 32; // Global warp index
    int lane = tid &(0xFFFF); // thread index within the warp
    //one warp per row
    double val; 
    int col;
    double sum[64] = {0};   
    if (nnz < 4096){
      int localCol;
      for (int i = tid; i < nnz; i+= blockDim.x){ 
	localCol = irp[startRow]+i;
	vals[i] = as[localCol];
	cols[i] = ja[localCol];
      }
    }
    //return;
    __syncthreads();
    if (warpId < col_multivector){
      for (int col_m = warpId; col_m < col_multivector; col_m +=32){
	for (int i = irp[startRow] + lane; i < irp[startRow+1]; i +=32){
	  if (nnz < 4096){
	    val = vals[i-irp[startRow]];
	    col = cols[i-irp[startRow]];
	  } else {
	    val = as[i];
	    col = ja[i];
	  }
	  sum[col_m] += val*multivector[col*col_multivector + col_m];     
	}
	sum[col_m] = warp_reduce(sum[col_m]);
	if (lane == 0){
	  if (beta == 0.0) {
	    resultData[startRow*col_multivector + col_m] = alpha*sum[col_m];   
	  } else {
	    resultData[startRow*col_multivector + col_m] = alpha*sum[col_m] +
	      beta*resultData[startRow*col_multivector + col_m];
	  }	  
	}
      }
    }
  }
}


__host__ int dCSGAMV(spgpuHandle_t handle, 
		     double beta,
		     double* y, 
		     double alpha, 
		     const double* as, 
		     const int* ja,
		     const int* irp,
		     int m,
		     int n,
		     int ncol,
		     int  numBlocks,
		     const int* rowBlocks,
		     const double *x,
		     int baseIndex,
		     int *rb)
{ 
  //  fprintf(stderr," dcsgamv  %d   \n",numBlocks);
  int maxBForACall = min(handle->maxGridSizeX, numBlocks);
  //int maxBForACall = 1024;
  int blockX = THREAD_BLOCK;
  int gridX = maxBForACall;
  int rp,rows, blcks, bp, numb;
  dim3 blockSize(blockX);
  dim3 gridSize(gridX);
  //fprintf(stderr," dcsgamv  start %d %d %d  \n",m,n,ncol);

  bp = 0;
  rp = 0;
  numb = numBlocks;  
  while (numb > maxBForACall) {//managing large vectors
    blcks = maxBForACall;
    rp = rb[bp]-baseIndex;
    rows = rb[bp+blcks]-rp;
    fprintf(stderr,"  rp %d  rows %d  bp %d  blcks %d\n",rp,rows,bp,blcks);
    dCSGAmvINNER<<<gridSize,blockSize>>>(alpha,as,ja,irp,x,rows,n,ncol,
					 &(rowBlocks[bp]),beta,&(y[rp]),baseIndex);
    bp   += blcks;
    numb -= blcks;
  }
  blcks = numb;
  rp = rb[bp]-baseIndex;
  rows = rb[bp+blcks]-rp;
  //fprintf(stderr,"  rp %d  rows %d  bp %d  blcks %d\n",rp,rows,bp,blcks);
  dCSGAmvINNER<<<gridSize,blockSize>>>(alpha,as,ja,irp,x,rows,n,ncol,
				       &(rowBlocks[bp]),beta,&(y[rp]),baseIndex);
  rp += rows;
  //fprintf(stderr,"  Final  rows %d  %d  \n",rows,rp);
  return(SPGPU_SUCCESS);
}
