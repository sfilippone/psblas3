/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2015
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
 
#include "stdio.h"
#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "vector.h"
}

#include "debug.h"

#define VALUE_TYPE hipFloatComplex
#define RES_VALUE_TYPE hipFloatComplex
#define TYPE_SYMBOL C
#include "abs_base.cuh"

