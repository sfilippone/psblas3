#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"
#include "hip/hip_complex.h"



extern "C"
{
#include "core.h"
#include "vector.h"
}


//#define USE_CUBLAS

#define BLOCK_SIZE 512

//#define ASSUME_LOCK_SYNC_PARALLELISM

static __device__ hipDoubleComplex ddotReductionResult[128];

__global__ void spgpuZdot_kern(int n, hipDoubleComplex* x, hipDoubleComplex* y)
{
	__shared__ hipDoubleComplex sSum[BLOCK_SIZE];

	hipDoubleComplex res = make_hipDoubleComplex(0.0, 0.0);

	hipDoubleComplex* lastX = x + n;

	x += threadIdx.x + blockIdx.x*BLOCK_SIZE;
	y += threadIdx.x + blockIdx.x*BLOCK_SIZE;

	int blockOffset = gridDim.x*BLOCK_SIZE;

	while (x < lastX)
    {
		res = hipCfma(x[0], y[0], res);
		
		x += blockOffset;
		y += blockOffset;

	}

	if (threadIdx.x >= 32)
		sSum[threadIdx.x] = res;

	__syncthreads();


	// Start reduction!

	if (threadIdx.x < 32) 
	{
		for (int i=1; i<BLOCK_SIZE/32; ++i)
		{
			res = hipCadd(res, sSum[i*32 + threadIdx.x]);
		}

	//useless (because inter-warp)
#ifndef	ASSUME_LOCK_SYNC_PARALLELISM
	}
	__syncthreads(); 

	if (threadIdx.x < 32) 
	{
#endif	

		hipDoubleComplex* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] = hipCadd(vsSum[threadIdx.x], vsSum[threadIdx.x + 16]);
		__syncthreads();
		if (threadIdx.x < 8) vsSum[threadIdx.x] = hipCadd(vsSum[threadIdx.x], vsSum[threadIdx.x + 8]);
		__syncthreads();
		if (threadIdx.x < 4) vsSum[threadIdx.x] = hipCadd(vsSum[threadIdx.x], vsSum[threadIdx.x + 4]);
		__syncthreads();
		if (threadIdx.x < 2) vsSum[threadIdx.x] = hipCadd(vsSum[threadIdx.x], vsSum[threadIdx.x + 2]);
		__syncthreads();
		if (threadIdx.x == 0)
		ddotReductionResult[blockIdx.x] = hipCadd(vsSum[0], vsSum[1]);

	}
}

hipDoubleComplex spgpuZdot(spgpuHandle_t handle, int n, __device hipDoubleComplex* a, __device hipDoubleComplex* b)
{
#ifdef USE_CUBLAS
	hipDoubleComplex res;
	hipblasDdot(n,x,1,y,1,&res);
	hipDeviceSynchronize();
	
	return res;
#else
	hipDoubleComplex res = make_hipDoubleComplex(0.0, 0.0);

#if 0 	
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device);	

	int blocks = min(128, min(prop.multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#else
	int blocks = min(128, min(handle->multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#endif
	
	hipDoubleComplex tRes[128];

	spgpuZdot_kern<<<blocks, BLOCK_SIZE, 0, handle->currentStream>>>(n, a, b);
	hipMemcpyFromSymbol(tRes, HIP_SYMBOL(ddotReductionResult),blocks*sizeof(hipDoubleComplex));

	for (int i=0; i<blocks; ++i)
	{
		res = hipCadd(res, tRes[i]);
	}

	cudaCheckError("CUDA error on ddot");
	
	return res;
#endif
}

void spgpuZmdot(spgpuHandle_t handle, hipDoubleComplex* y, int n, __device hipDoubleComplex* a, __device hipDoubleComplex* b, int count, int pitch)
{
	for (int i=0; i<count; ++i)
	{
		y[i] = spgpuZdot(handle, n, a, b);
		a += pitch;
		b += pitch;
	}
}
