#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2014
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cudadebug.h"
#include "cudalang.h"
#include "hip/hip_complex.h"


extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512

#if 1

__global__ void spgpuCaxpby_krn(hipFloatComplex *z, int n, hipFloatComplex beta, hipFloatComplex *y, hipFloatComplex alpha, hipFloatComplex* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (cuFloatComplex_isZero(beta)) {
	  for ( ; id < n; id +=gridSize)
	    //if (id,n) 
	    {
	      // Since z, x and y are accessed with the same offset by the same thread,
	      // and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      
	      z[id] = hipCmulf(alpha,x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    //if (id,n) 
	    {
	      z[id] = hipCfmaf(beta, y[id], hipCmulf(alpha, x[id]));
	    }
	}
}

void spgpuCaxpby(spgpuHandle_t handle,
	__device hipFloatComplex *z,
	int n,
	hipFloatComplex beta,
	__device hipFloatComplex *y,
	hipFloatComplex alpha,
	__device hipFloatComplex* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuCaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

#else


__global__ void spgpuCaxpby_krn(hipFloatComplex *z, int n, hipFloatComplex beta, hipFloatComplex *y, hipFloatComplex alpha, hipFloatComplex* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (cuFloatComplex_isZero(beta))
			z[id] = hipCmulf(alpha,x[id]);
		else
			z[id] = hipCfmaf(beta, y[id], hipCmulf(alpha, x[id]));
	}
}


void spgpuCaxpby_(spgpuHandle_t handle,
	__device hipFloatComplex *z,
	int n,
	hipFloatComplex beta,
	__device hipFloatComplex *y,
	hipFloatComplex alpha,
	__device hipFloatComplex* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuCaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuCaxpby(spgpuHandle_t handle,
	__device hipFloatComplex *z,
	int n,
	hipFloatComplex beta,
	__device hipFloatComplex *y,
	hipFloatComplex alpha,
	__device hipFloatComplex* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);

	while (n > maxNForACall) //managing large vectors
	{
		spgpuCaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuCaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on saxpby");
}
#endif
void spgpuCmaxpby(spgpuHandle_t handle,
		  __device hipFloatComplex *z,
		  int n,
		  hipFloatComplex beta,
		  __device hipFloatComplex *y,
		  hipFloatComplex alpha,
		  __device hipFloatComplex* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuCaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
