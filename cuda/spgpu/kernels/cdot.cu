#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"
#include "hip/hip_complex.h"


extern "C"
{
#include "core.h"
#include "vector.h"
}

//#define USE_CUBLAS

#define BLOCK_SIZE 320
//#define BLOCK_SIZE 512

//#define ASSUME_LOCK_SYNC_PARALLELISM


static __device__ hipFloatComplex sdotReductionResult[128];

__global__ void spgpuCdot_kern(int n, hipFloatComplex* x, hipFloatComplex* y)
{
	__shared__ hipFloatComplex sSum[BLOCK_SIZE];

	hipFloatComplex res = make_hipFloatComplex(0.0f, 0.0f);

	hipFloatComplex* lastX = x + n;

	x += threadIdx.x + blockIdx.x*BLOCK_SIZE;
	y += threadIdx.x + blockIdx.x*BLOCK_SIZE;

	int blockOffset = gridDim.x*BLOCK_SIZE;

	int numSteps = (lastX - x + blockOffset - 1)/blockOffset;

	// prefetching
	for (int j = 0; j < numSteps / 2; j++)
    {
		hipFloatComplex x1 = x[0]; x += blockOffset;
		hipFloatComplex y1 = y[0]; y += blockOffset;
		hipFloatComplex x2 = x[0]; x += blockOffset;
		hipFloatComplex y2 = y[0]; y += blockOffset;

		res = hipCfmaf(x1, y1, res);
		res = hipCfmaf(x2, y2, res);

	}

	if (numSteps % 2)
	{
		res = hipCfmaf(*x, *y, res);
	}

	if (threadIdx.x >= 32)
		sSum[threadIdx.x] = res;

	__syncthreads();


	// Start reduction!

	if (threadIdx.x < 32) 
	{
		for (int i=1; i<BLOCK_SIZE/32; ++i)
		{
			res = hipCaddf(res, sSum[i*32 + threadIdx.x]);
		}

	//useless (because inter-warp)
#ifndef	ASSUME_LOCK_SYNC_PARALLELISM
	}
	__syncthreads(); 

	if (threadIdx.x < 32) 
	{
#endif	

		hipFloatComplex* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] = hipCaddf(vsSum[threadIdx.x], vsSum[threadIdx.x + 16]);
		__syncthreads();
		if (threadIdx.x < 8) vsSum[threadIdx.x] = hipCaddf(vsSum[threadIdx.x], vsSum[threadIdx.x + 8]);
		__syncthreads();
		if (threadIdx.x < 4) vsSum[threadIdx.x] = hipCaddf(vsSum[threadIdx.x], vsSum[threadIdx.x + 4]);
		__syncthreads();
		if (threadIdx.x < 2) vsSum[threadIdx.x] = hipCaddf(vsSum[threadIdx.x], vsSum[threadIdx.x + 2]);
		__syncthreads();
		if (threadIdx.x == 0)
			sdotReductionResult[blockIdx.x] = hipCaddf(vsSum[0], vsSum[1]);
	}
}

hipFloatComplex spgpuCdot(spgpuHandle_t handle, int n, __device hipFloatComplex* a, __device hipFloatComplex* b)
{

#ifdef USE_CUBLAS
	hipFloatComplex res;
	hipblasSdot(n,x,1,y,1,&res);
	hipDeviceSynchronize();
	
	return res;
#else
	hipFloatComplex res = make_hipFloatComplex(0.0f, 0.0f);

#if 0 	
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device);	

	int blocks = min(128, min(prop.multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#else
	int blocks = min(128, min(handle->multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#endif

	
	hipFloatComplex tRes[128];

	spgpuCdot_kern<<<blocks, (BLOCK_SIZE), 0, handle->currentStream>>>(n, a, b);
	hipMemcpyFromSymbol(tRes, HIP_SYMBOL(sdotReductionResult), blocks*sizeof(hipFloatComplex));

	for (int i=0; i<blocks; ++i)
	{
		res = hipCaddf(res, tRes[i]);
	}

	cudaCheckError("CUDA error on sdot (blocks: %i, regs per block: %i)\n", blocks, prop.regsPerBlock);
	
	return res;
#endif
}

void spgpuCmdot(spgpuHandle_t handle, hipFloatComplex* y, int n, __device hipFloatComplex* a, __device hipFloatComplex* b, int count, int pitch)
{
	for (int i=0; i<count; ++i)
	{
		y[i] = spgpuCdot(handle, n, a, b);
		a += pitch;
		b += pitch;
	}
}
