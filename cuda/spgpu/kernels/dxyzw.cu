#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuDxyzw_krn(int n, double  a, double  b,
			       double  c, double  d,
			       double  e, double  f,
			       double * x, double  *y,
			       double  *z, double  *w)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	double  ty, tz;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  ty    = PREC_DADD(PREC_DADD(a, x[id]), PREC_DMUL(b,y[id]));
	  tz    = PREC_DADD(PREC_DADD(c, ty), PREC_DMUL(d,z[id]));
	  w[id] = PREC_DADD(PREC_DADD(e, tz), PREC_DMUL(f,w[id]));
	  y[id] = ty;
	  z[id] = tz;
	}
}


void spgpuDxyzw(spgpuHandle_t handle,
		int n,
		double  a, double  b,
		double  c, double  d,
		double  e, double  f,
		__device double * x,
		__device double * y,
		__device double * z,
		__device double *w)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuDxyzw_krn<<<grid, block, 0, handle->currentStream>>>(n, a,b,c,d,e,f,
								  x, y, z,w);
}

