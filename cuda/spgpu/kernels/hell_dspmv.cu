#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2014
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <stdio.h>
extern "C"
{
#include "core.h"
#include "hell.h"
  int getGPUSharedMemPerBlock();
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}

#include "debug.h"

#define VALUE_TYPE double
#define TYPE_SYMBOL D
#define TEX_FETCH_TYPE int2
#include "hell_spmv_base.cuh"



#if 0

#define MMBSZ 8

#undef GEN_SPGPU_HELL_NAME
#define GEN_SPGPU_HELL_NAME(x) CONCAT(CONCAT(spgpu,x),hellspmm)
#undef GEN_SPGPU_HELL_NAME_VANILLA
#define GEN_SPGPU_HELL_NAME_VANILLA(x) CONCAT(CONCAT(spgpu,x),hellspmm_vanilla)


__global__ void
CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn)
  (int count, VALUE_TYPE *z, int zPitch, const VALUE_TYPE *y, int yPitch,
   VALUE_TYPE alpha, const VALUE_TYPE* cM, const int* rP,
   int hackSize, const int* hackOffsets, const int* rS, int rows,
   const VALUE_TYPE *x, int xPitch, 
   VALUE_TYPE beta, int baseIndex)
{
  VALUE_TYPE *pz,*px,*py;
  VALUE_TYPE zProd = CONCAT(zero_,VALUE_TYPE)();
  VALUE_TYPE yVal; 
  __shared__ VALUE_TYPE temp[MMBSZ][THREAD_BLOCK];

  int *rrP;
  VALUE_TYPE *rcM;
  
  unsigned int i = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int gridSize =  gridDim.x * blockDim.x;
  
  while (i < rows) {
    int j; 
    int hackId = i / hackSize;
    int hackLaneId = i % hackSize;
    
    int hackOffset;
    unsigned int laneId = threadIdx.x % 32;
    if (laneId == 0)
      hackOffset = hackOffsets[hackId];
    //__syncthreads();
    hackOffset = __shfl_sync(0xFFFFFFFF,hackOffset, 0) + hackLaneId;		
    
    rrP = (int *) rP + hackOffset; 
    rcM = (VALUE_TYPE *) cM + hackOffset; 
    
    int rowSize = rS[i];
    for (int k=0; k<count; k++) {
      temp[k][threadIdx.x] = CONCAT(zero_,VALUE_TYPE)();
    }      
    
    for (int j = 0; j < rowSize; j++) {
      int pointer;
      VALUE_TYPE value;
      VALUE_TYPE fetch;
      
      pointer = rrP[0] - baseIndex;
      rrP += hackSize;
      
      value = rcM[0];
      rcM += hackSize;

      px = (VALUE_TYPE *) x;
      for (int k=0; k<count; k++) {
	fetch = px[pointer]; 
	temp[k][threadIdx.x] =
	  CONCAT(VALUE_TYPE, _fma)(value, fetch, temp[k][threadIdx.x]);
	px = px + xPitch;
      }
    }
    // Since z and y are accessed with the same offset by the same thread,
    // and the write to z follows the y read, y and z can share the same base address (in-place computing).
    py = (VALUE_TYPE *) y;
    pz = z;
    if (CONCAT(VALUE_TYPE, _isNotZero(beta))) {
      for (int k=0; k<count; k++) {
	yVal = py[i];
	pz[i] = CONCAT(VALUE_TYPE, _fma)(beta,
					 yVal, CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]));
	py += yPitch;
	pz += zPitch;
      }
    } else {
      for (int k=0; k<count; k++) {
	pz[i] = CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]);
	pz += zPitch;
      }
    }
    
    i += gridSize;
  }
}


void
GEN_SPGPU_HELL_NAME(TYPE_SYMBOL)
     (spgpuHandle_t handle,
        int count, 
        VALUE_TYPE* z,
        int zPitch,
	const VALUE_TYPE *y,
        int  yPitch,
	VALUE_TYPE alpha, 
	const VALUE_TYPE* cM, 
	const int* rP, 
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const VALUE_TYPE *x,
        int xPitch,
	VALUE_TYPE beta, 
      int baseIndex)
{
  VALUE_TYPE *px,*py, *pz;
  int cnt, c1;

  dim3 block (THREAD_BLOCK, 1);
  //  dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);
  // Should we generalize the code to 1/2/4/8 threads per row?
  // And maybe adjust THREAD_BLOCK size? 
  int shrMemSize,maxShmemSz;
  int numMp=getGPUMultiProcessors();
  int maxThMp=getGPUMaxThreadsPerMP();
  int nmblksMp=maxThMp/THREAD_BLOCK;
  int nmblk=nmblksMp*numMp;
  dim3 grid (nmblk);

  maxShmemSz=getGPUSharedMemPerBlock();
  shrMemSize=MMBSZ*THREAD_BLOCK*sizeof(VALUE_TYPE);
  if (shrMemSize > maxShmemSz) {
    fprintf(stderr,"Fatal error: SHMEM size too large %ld %ld\n",shrMemSize,maxShmemSz);
    return;
  }
  cnt = count;
  px = (VALUE_TYPE *) x;
  py = (VALUE_TYPE *) y;
  pz = (VALUE_TYPE *) z;	  
  while (cnt > 2*MMBSZ) {
    CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn) 
      <<< grid, block, shrMemSize, handle->currentStream >>> (MMBSZ, pz, zPitch,py, yPitch,
							      alpha, cM, rP, hackSize, hackOffsets,
							      rS, rows, px, xPitch, beta, baseIndex);
    px += xPitch*MMBSZ;
    py += yPitch*MMBSZ;
    pz += zPitch*MMBSZ;
    cnt -= MMBSZ;
  }
  if (cnt > MMBSZ) {
    c1 = cnt/2;
    CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn) 
      <<< grid, block, shrMemSize, handle->currentStream >>> (c1, pz, zPitch,py, yPitch,
							      alpha, cM, rP, hackSize, hackOffsets,
							      rS, rows, px, xPitch, beta, baseIndex);
    cnt -= c1;
  }
  if (cnt > MMBSZ) {
    fprintf(stderr,"Invalid residual count %d\n",cnt);
  } else if (cnt > 0){
    CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn) 
      <<< grid, block, shrMemSize, handle->currentStream >>> (cnt, pz, zPitch,py, yPitch,
							      alpha, cM, rP, hackSize, hackOffsets,
							      rS, rows, px, xPitch, beta, baseIndex);
  }
  cudaCheckError("CUDA error on hell_spmm");
}

#elif defined(NEW_MM)

#define MMBSZ 8

#undef GEN_SPGPU_HELL_NAME
#define GEN_SPGPU_HELL_NAME(x) CONCAT(CONCAT(spgpu,x),hellspmm)
#undef GEN_SPGPU_HELL_NAME_VANILLA
#define GEN_SPGPU_HELL_NAME_VANILLA(x) CONCAT(CONCAT(spgpu,x),hellspmm_vanilla)


__global__ void
CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn)
  (int count, VALUE_TYPE *z, int zPitch, const VALUE_TYPE *y, int yPitch,
   VALUE_TYPE alpha, const VALUE_TYPE* cM, const int* rP,
   int hackSize, const int* hackOffsets, const int* rS, int rows,
   const VALUE_TYPE *x, int xPitch, 
   VALUE_TYPE beta, int baseIndex)
{
  VALUE_TYPE *pz,*px,*py;
  VALUE_TYPE zProd = CONCAT(zero_,VALUE_TYPE)();
  VALUE_TYPE yVal; 
  __shared__ VALUE_TYPE temp[MMBSZ][THREAD_BLOCK];
        
  int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
    
  if (i < rows) {
    int j; 
    int hackId = i / hackSize;
    int hackLaneId = i % hackSize;
    
    int hackOffset;
    unsigned int laneId = threadIdx.x % 32;
    if (laneId == 0)
      hackOffset = hackOffsets[hackId];
    //__syncthreads();
    hackOffset = __shfl_sync(0xFFFFFFFF,hackOffset, 0) + hackLaneId;		
    
    rP += hackOffset; 
    cM += hackOffset; 
    
    int rowSize = rS[i];
    for (int k=0; k<count; k++) {
      temp[k][threadIdx.x] = CONCAT(zero_,VALUE_TYPE)();
    }      
    
    for (int j = 0; j < rowSize; j++) {
      int pointer;
      VALUE_TYPE value;
      VALUE_TYPE fetch;
      
      pointer = rP[0] - baseIndex;
      rP += hackSize;
      
      value = cM[0];
      cM += hackSize;

      px = (VALUE_TYPE *) x;
      for (int k=0; k<count; k++) {
	fetch = px[pointer]; 
	temp[k][threadIdx.x] =
	  CONCAT(VALUE_TYPE, _fma)(value, fetch, temp[k][threadIdx.x]);
	px = px + xPitch;
      }
    }
    // Since z and y are accessed with the same offset by the same thread,
    // and the write to z follows the y read, y and z can share the same base address (in-place computing).
    py = (VALUE_TYPE *) y;
    pz = z;
    if (CONCAT(VALUE_TYPE, _isNotZero(beta)))
      for (int k=0; k<count; k++) {
	yVal = py[i];
	pz[i] = CONCAT(VALUE_TYPE, _fma)(beta, yVal, CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]));
	py += yPitch;
	pz += zPitch;
      }
    else
      for (int k=0; k<count; k++) {
	pz[i] = CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]);
	pz += zPitch;
      }
  }
}


void
CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL))
  (spgpuHandle_t handle, int count, VALUE_TYPE* z, int zPitch, const VALUE_TYPE *y, int yPitch,
   VALUE_TYPE alpha, const VALUE_TYPE* cM, const int* rP, int hackSize, const int* hackOffsets,
   const int* rS,  const __device int* rIdx, int rows,
   const VALUE_TYPE *x, int xPitch, VALUE_TYPE beta, int baseIndex)
{
  dim3 block (THREAD_BLOCK, 1);
  dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);
  // Should we generalize the code to 1/2/4/8 threads per row?
  // And maybe adjust THREAD_BLOCK size? 
  int shrMemSize,maxShmemSz;
  maxShmemSz=getGPUSharedMemPerBlock();
  shrMemSize=MMBSZ*THREAD_BLOCK*sizeof(VALUE_TYPE);
  CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn) 
    <<< grid, block, shrMemSize, handle->currentStream >>> (count, z, zPitch,y, yPitch,
							    alpha, cM, rP, hackSize, hackOffsets, rS, rows,
							    x, xPitch, beta, baseIndex);
}

void
GEN_SPGPU_HELL_NAME(TYPE_SYMBOL)
     (spgpuHandle_t handle,
        int count, 
        VALUE_TYPE* z,
        int zPitch,
	const VALUE_TYPE *y,
        int  yPitch,
	VALUE_TYPE alpha, 
	const VALUE_TYPE* cM, 
	const int* rP, 
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const VALUE_TYPE *x,
        int xPitch,
	VALUE_TYPE beta, 
      int baseIndex)
{
  VALUE_TYPE *px,*py, *pz;
  int cnt;
  int maxNForACall = max(handle->maxGridSizeX, THREAD_BLOCK*handle->maxGridSizeX);
  
  // maxNForACall should be a multiple of hackSize
  maxNForACall = (maxNForACall/hackSize)*hackSize;
  int maxShmemSz;
  maxShmemSz=getGPUSharedMemPerBlock();
  //fprintf(stderr,"MaxSHmemSz  %d \n",maxShmemSz);
  while (rows > maxNForACall) {//managing large vectors
    cnt = count;
    px = (VALUE_TYPE *) x;
    py = (VALUE_TYPE *) y;
    pz = (VALUE_TYPE *) z;	  
    while (cnt > MMBSZ) {
      //fprintf(stderr,"counts %d %d %d :  pointers: %p %p %p\n",rows,cnt,MMBSZ,px,py,pz);    
      CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, MMBSZ, pz, zPitch,
							  py, yPitch,
							  alpha, cM, rP,
							  hackSize, hackOffsets,
							  rS, rIdx,
							  maxNForACall,
							  px, xPitch, beta, baseIndex);
      px += xPitch*MMBSZ;
      py += yPitch*MMBSZ;
      pz += zPitch*MMBSZ;
      cnt -= MMBSZ;
    }
    if (cnt >0) {
      CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, cnt, pz, zPitch,
							  py, yPitch,
							  alpha, cM, rP,
							  hackSize, hackOffsets,
							  rS, rIdx,
							  maxNForACall,
							  px, xPitch, beta, baseIndex);
    }

    y = y + maxNForACall;
    z = z + maxNForACall;
    hackOffsets = hackOffsets + maxNForACall/hackSize;
    rS = rS + maxNForACall;
    
    rows -= maxNForACall;
  }
  cnt = count;
  px = (VALUE_TYPE *) x;
  py = (VALUE_TYPE *) y;
  pz = (VALUE_TYPE *) z;	  
  while (cnt > MMBSZ) {
    //fprintf(stderr,"counts %d %d %d :  pointers: %p %p %p\n",rows,cnt,MMBSZ,px,py,pz);    
    CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, MMBSZ, pz, zPitch, py, yPitch,
							alpha, cM, rP, hackSize, hackOffsets,
							rS, rIdx, rows,
							px, xPitch, beta, baseIndex);
    px += xPitch*MMBSZ;
    py += yPitch*MMBSZ;
    pz += zPitch*MMBSZ;
    cnt -= MMBSZ;
  }
  if (cnt >0) {
    CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, cnt, pz, zPitch,
							py, yPitch,
							alpha, cM, rP,
							hackSize, hackOffsets,
							rS, rIdx,
							rows,
							px, xPitch, beta, baseIndex);
  }
  
  
  cudaCheckError("CUDA error on hell_spmm");
}


#endif

