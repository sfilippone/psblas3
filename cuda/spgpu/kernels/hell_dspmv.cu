#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2014
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <stdio.h>
extern "C"
{
#include "core.h"
#include "hell.h"
  int  getGPUSharedMemPerBlock();
}

#include "debug.h"

#define VALUE_TYPE double
#define TYPE_SYMBOL D
#define TEX_FETCH_TYPE int2
#include "hell_spmv_base.cuh"



#if defined(NEW_MM)

#define MMBSZ 8

#undef GEN_SPGPU_HELL_NAME
#define GEN_SPGPU_HELL_NAME(x) CONCAT(CONCAT(spgpu,x),hellspmm)
#undef GEN_SPGPU_HELL_NAME_VANILLA
#define GEN_SPGPU_HELL_NAME_VANILLA(x) CONCAT(CONCAT(spgpu,x),hellspmm_vanilla)


__global__ void
CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn)
  (int count, VALUE_TYPE *z, int zPitch, const VALUE_TYPE *y, int yPitch,
   VALUE_TYPE alpha, const VALUE_TYPE* cM, const int* rP,
   int hackSize, const int* hackOffsets, const int* rS, int rows,
   const VALUE_TYPE *x, int xPitch, 
   VALUE_TYPE beta, int baseIndex)
{
  VALUE_TYPE *pz,*px,*py;
  VALUE_TYPE zProd = CONCAT(zero_,VALUE_TYPE)();
  VALUE_TYPE yVal; 
  __shared__ VALUE_TYPE temp[MMBSZ][THREAD_BLOCK];
        
  int i = threadIdx.x + blockIdx.x * (THREAD_BLOCK);
    
  if (i < rows) {
    int j; 
    int hackId = i / hackSize;
    int hackLaneId = i % hackSize;
    
    int hackOffset;
    unsigned int laneId = threadIdx.x % 32;
    if (laneId == 0)
      hackOffset = hackOffsets[hackId];
    //__syncthreads();
    hackOffset = __shfl_sync(0xFFFFFFFF,hackOffset, 0) + hackLaneId;		
    
    rP += hackOffset; 
    cM += hackOffset; 
    
    int rowSize = rS[i];
    for (int k=0; k<count; k++) {
      temp[k][threadIdx.x] = CONCAT(zero_,VALUE_TYPE)();
    }      
    
    for (int j = 0; j < rowSize; j++) {
      int pointer;
      VALUE_TYPE value;
      VALUE_TYPE fetch;
      
      pointer = rP[0] - baseIndex;
      rP += hackSize;
      
      value = cM[0];
      cM += hackSize;

      px = (VALUE_TYPE *) x;
      for (int k=0; k<count; k++) {
	fetch = px[pointer]; 
	temp[k][threadIdx.x] =
	  CONCAT(VALUE_TYPE, _fma)(value, fetch, temp[k][threadIdx.x]);
	px = px + xPitch;
      }
    }
    // Since z and y are accessed with the same offset by the same thread,
    // and the write to z follows the y read, y and z can share the same base address (in-place computing).
    py = (VALUE_TYPE *) y;
    pz = z;
    if (CONCAT(VALUE_TYPE, _isNotZero(beta)))
      for (int k=0; k<count; k++) {
	yVal = py[i];
	pz[i] = CONCAT(VALUE_TYPE, _fma)(beta, yVal, CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]));
	py += yPitch;
	pz += zPitch;
      }
    else
      for (int k=0; k<count; k++) {
	pz[i] = CONCAT(VALUE_TYPE, _mul) (alpha,  temp[k][threadIdx.x]);
	pz += zPitch;
      }
  }
}


void
CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL))
  (spgpuHandle_t handle, int count, VALUE_TYPE* z, int zPitch, const VALUE_TYPE *y, int yPitch,
   VALUE_TYPE alpha, const VALUE_TYPE* cM, const int* rP, int hackSize, const int* hackOffsets,
   const int* rS,  const __device int* rIdx, int rows,
   const VALUE_TYPE *x, int xPitch, VALUE_TYPE beta, int baseIndex)
{
  dim3 block (THREAD_BLOCK, 1);
  dim3 grid ((rows + THREAD_BLOCK - 1) / THREAD_BLOCK);
  // Should we generalize the code to 1/2/4/8 threads per row?
  // And maybe adjust THREAD_BLOCK size? 
  int shrMemSize,maxShmemSz;
  maxShmemSz=getGPUSharedMemPerBlock();
  shrMemSize=MMBSZ*THREAD_BLOCK*sizeof(VALUE_TYPE);
  CONCAT(GEN_SPGPU_HELL_NAME(TYPE_SYMBOL), _krn) 
    <<< grid, block, shrMemSize, handle->currentStream >>> (count, z, zPitch,y, yPitch,
							    alpha, cM, rP, hackSize, hackOffsets, rS, rows,
							    x, xPitch, beta, baseIndex);
}

void
GEN_SPGPU_HELL_NAME(TYPE_SYMBOL)
     (spgpuHandle_t handle,
        int count, 
        VALUE_TYPE* z,
        int zPitch,
	const VALUE_TYPE *y,
        int  yPitch,
	VALUE_TYPE alpha, 
	const VALUE_TYPE* cM, 
	const int* rP, 
	int hackSize,
	const __device int* hackOffsets, 
	const __device int* rS,
	const __device int* rIdx, 
	int rows, 
	const VALUE_TYPE *x,
        int xPitch,
	VALUE_TYPE beta, 
      int baseIndex)
{
  VALUE_TYPE *px,*py, *pz;
  int cnt;
  int maxNForACall = max(handle->maxGridSizeX, THREAD_BLOCK*handle->maxGridSizeX);
  
  // maxNForACall should be a multiple of hackSize
  maxNForACall = (maxNForACall/hackSize)*hackSize;
  int maxShmemSz;
  maxShmemSz=getGPUSharedMemPerBlock();
  //fprintf(stderr,"MaxSHmemSz  %d \n",maxShmemSz);
  while (rows > maxNForACall) {//managing large vectors
    cnt = count;
    px = (VALUE_TYPE *) x;
    py = (VALUE_TYPE *) y;
    pz = (VALUE_TYPE *) z;	  
    while (cnt > MMBSZ) {
      //fprintf(stderr,"counts %d %d %d :  pointers: %p %p %p\n",rows,cnt,MMBSZ,px,py,pz);    
      CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, MMBSZ, pz, zPitch,
							  py, yPitch,
							  alpha, cM, rP,
							  hackSize, hackOffsets,
							  rS, rIdx,
							  maxNForACall,
							  px, xPitch, beta, baseIndex);
      px += xPitch*MMBSZ;
      py += yPitch*MMBSZ;
      pz += zPitch*MMBSZ;
      cnt -= MMBSZ;
    }
    if (cnt >0) {
      CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, cnt, pz, zPitch,
							  py, yPitch,
							  alpha, cM, rP,
							  hackSize, hackOffsets,
							  rS, rIdx,
							  maxNForACall,
							  px, xPitch, beta, baseIndex);
    }

    y = y + maxNForACall;
    z = z + maxNForACall;
    hackOffsets = hackOffsets + maxNForACall/hackSize;
    rS = rS + maxNForACall;
    
    rows -= maxNForACall;
  }
  cnt = count;
  px = (VALUE_TYPE *) x;
  py = (VALUE_TYPE *) y;
  pz = (VALUE_TYPE *) z;	  
  while (cnt > MMBSZ) {
    //fprintf(stderr,"counts %d %d %d :  pointers: %p %p %p\n",rows,cnt,MMBSZ,px,py,pz);    
    CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, MMBSZ, pz, zPitch, py, yPitch,
							alpha, cM, rP, hackSize, hackOffsets,
							rS, rIdx, rows,
							px, xPitch, beta, baseIndex);
    px += xPitch*MMBSZ;
    py += yPitch*MMBSZ;
    pz += zPitch*MMBSZ;
    cnt -= MMBSZ;
  }
  if (cnt >0) {
    CONCAT(_,GEN_SPGPU_HELL_NAME_VANILLA(TYPE_SYMBOL)) (handle, cnt, pz, zPitch,
							py, yPitch,
							alpha, cM, rP,
							hackSize, hackOffsets,
							rS, rIdx,
							rows,
							px, xPitch, beta, baseIndex);
  }
  
  
  cudaCheckError("CUDA error on hell_spmm");
}

#endif

