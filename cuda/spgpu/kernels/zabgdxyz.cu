#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuZabgdxyz_krn(int n, hipDoubleComplex  alpha, hipDoubleComplex  beta,
				  hipDoubleComplex  gamma, hipDoubleComplex  delta,
				  hipDoubleComplex * x, hipDoubleComplex  *y, hipDoubleComplex  *z)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	hipDoubleComplex  t;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  if (cuDoubleComplex_isZero(beta)) 
	    t = hipCmul(alpha,x[id]);
	  else
	    t = hipCfma(alpha, x[id], hipCmul(beta,y[id]));
	  if (cuDoubleComplex_isZero(delta))
	    z[id] = hipCmul(gamma, t);
	  else
	    z[id] = hipCfma(gamma, t, hipCmul(delta,z[id]));
	  y[id] = t;
	}
}


void spgpuZabgdxyz(spgpuHandle_t handle,
		   int n,
		   hipDoubleComplex  alpha,
		   hipDoubleComplex  beta,
		   hipDoubleComplex  gamma,
		   hipDoubleComplex  delta,
		   __device hipDoubleComplex * x,
		   __device hipDoubleComplex * y,
		   __device hipDoubleComplex  *z)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuZabgdxyz_krn<<<grid, block, 0, handle->currentStream>>>(n, alpha, beta, gamma, delta,
								   x, y, z);
}

