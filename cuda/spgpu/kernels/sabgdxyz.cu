#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuSabgdxyz_krn(int n, float alpha, float beta, float gamma, float delta,
				  float* x, float *y, float *z)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	float t;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  if (beta == 0.0)
	    t = PREC_FMUL(alpha,x[id]);
	  else
	    t = PREC_FADD(PREC_FMUL(alpha, x[id]), PREC_FMUL(beta,y[id]));
	  if (delta == 0.0)
	    z[id] = gamma * t;
	  else
	    z[id] = PREC_FADD(PREC_FMUL(gamma, t), PREC_FMUL(delta,z[id]));
	  y[id] = t;
	}
}


void spgpuSabgdxyz(spgpuHandle_t handle,
		   int n,
		   float alpha,
		   float beta,
		   float gamma,
		   float delta,
		   __device float* x,
		   __device float* y,
		   __device float *z)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuSabgdxyz_krn<<<grid, block, 0, handle->currentStream>>>(n, alpha, beta, gamma, delta,
								   x, y, z);
}

