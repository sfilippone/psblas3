#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */
#include "cudadebug.h"
#include "cudalang.h"

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512


#if 1
__global__ void spgpuSaxpby_krn(float *z, int n, float beta, float *y, float alpha, float* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (beta == 0.0f) {
	  for ( ; id < n; id +=gridSize)
	    {
	      // Since z, x and y are accessed with the same offset by the same thread,
	      // and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      
	      z[id] = PREC_FMUL(alpha,x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    {
	      // Since z, x and y are accessed with the same offset by the same thread,
	      // and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).
	      z[id] = PREC_FADD(PREC_FMUL(alpha, x[id]), PREC_FMUL(beta,y[id]));
	    }
	}
}

void spgpuSaxpby(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuSaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

#else

__global__ void spgpuSaxpby_krn(float *z, int n, float beta, float *y, float alpha, float* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0f)
			z[id] = PREC_FMUL(alpha,x[id]);
		else
			z[id] = PREC_FADD(PREC_FMUL(alpha, x[id]), PREC_FMUL(beta,y[id]));
	}
}



void spgpuSaxpby_(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device float* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuSaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuSaxpby(spgpuHandle_t handle,
	__device float *z,
	int n,
	float beta,
	__device float *y,
	float alpha,
	__device float* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpuSaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuSaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on saxpby");
}
#endif
void spgpuSmaxpby(spgpuHandle_t handle,
		  __device float *z,
		  int n,
		  float beta,
		  __device float *y,
		  float alpha,
		  __device float* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuSaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
