#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "stdio.h"
#include "cudalang.h"
#include "cudadebug.h"


extern "C"
{
#include "core.h"
#include "vector.h"
}


//#define USE_CUBLAS

#define BLOCK_SIZE 512

//#define ASSUME_LOCK_SYNC_PARALLELISM

static __device__ double ddotReductionResult[128];

__global__ void spgpuDdot_kern(int n, double* x, double* y)
{
	__shared__ double sSum[BLOCK_SIZE];

	double res = 0;

	double* lastX = x + n;

	x += threadIdx.x + blockIdx.x*BLOCK_SIZE;
	y += threadIdx.x + blockIdx.x*BLOCK_SIZE;

	int blockOffset = gridDim.x*BLOCK_SIZE;

	while (x < lastX)
    {
		res = PREC_DADD(res, PREC_DMUL(x[0], y[0]));
		
		x += blockOffset;
		y += blockOffset;

	}

	if (threadIdx.x >= 32)
		sSum[threadIdx.x] = res;

	__syncthreads();


	// Start reduction!

	if (threadIdx.x < 32) 
	{
		for (int i=1; i<BLOCK_SIZE/32; ++i)
		{
			res += sSum[i*32 + threadIdx.x];
		}

	//useless (because inter-warp)
#ifndef	ASSUME_LOCK_SYNC_PARALLELISM
	}
	__syncthreads(); 

	if (threadIdx.x < 32) 
	{
#endif	

#ifdef ASSUME_LOCK_SYNC_PARALLELISM
		volatile double* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] += vsSum[threadIdx.x + 16];
		if (threadIdx.x < 8) vsSum[threadIdx.x] += vsSum[threadIdx.x + 8];
		if (threadIdx.x < 4) vsSum[threadIdx.x] += vsSum[threadIdx.x + 4];
		if (threadIdx.x < 2) vsSum[threadIdx.x] += vsSum[threadIdx.x + 2];
		if (threadIdx.x == 0)
			ddotReductionResult[blockIdx.x] = vsSum[0] + vsSum[1];

#else
		double* vsSum = sSum;
		vsSum[threadIdx.x] = res;

		if (threadIdx.x < 16) vsSum[threadIdx.x] += vsSum[threadIdx.x + 16];
		__syncthreads();
		if (threadIdx.x < 8) vsSum[threadIdx.x] += vsSum[threadIdx.x + 8];
		__syncthreads();
		if (threadIdx.x < 4) vsSum[threadIdx.x] += vsSum[threadIdx.x + 4];
		__syncthreads();
		if (threadIdx.x < 2) vsSum[threadIdx.x] += vsSum[threadIdx.x + 2];
		__syncthreads();
		if (threadIdx.x == 0)
		ddotReductionResult[blockIdx.x] = vsSum[0] + vsSum[1];
#endif
	}
}

double spgpuDdot(spgpuHandle_t handle, int n, __device double* a, __device double* b)
{
#ifdef USE_CUBLAS
	double res;
	hipblasDdot(n,x,1,y,1,&res);
	hipDeviceSynchronize();
	
	return res;
#else
	double res = 0;

	int device;
	hipGetDevice(&device);
#if 0 	
	int device;
	hipGetDevice(&device);
	struct hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop,device);	

	int blocks = min(128, min(prop.multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#else
	int blocks = min(128, min(handle->multiProcessorCount, (n+BLOCK_SIZE-1)/BLOCK_SIZE));
#endif
	
	double tRes[128];

	spgpuDdot_kern<<<blocks, BLOCK_SIZE, 0, handle->currentStream>>>(n, a, b);
	hipMemcpyFromSymbol(tRes, HIP_SYMBOL(ddotReductionResult),blocks*sizeof(double));

	for (int i=0; i<blocks; ++i)
	{
		res += tRes[i];
	}

	cudaCheckError("CUDA error on ddot");
	
	return res;
#endif
}

void spgpuDmdot(spgpuHandle_t handle, double* y, int n, __device double* a, __device double* b, int count, int pitch)
{
	for (int i=0; i<count; ++i)
	{
		y[i] = spgpuDdot(handle, n, a, b);
		a += pitch;
		b += pitch;
	}
}
