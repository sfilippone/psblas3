#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
  //#include "cuda_util.h"
}


#include "debug.h"

#define BLOCK_SIZE 512


#if 1
__global__ void spgpuDaxpby_krn(double *z, int n, double beta, double *y, double alpha, double* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	if (beta == 0.0) {
	  for ( ; id < n; id +=gridSize)
	    {
	      
	      z[id] = PREC_DMUL(alpha,x[id]);
	    }
	} else {
	  for ( ; id < n; id +=gridSize)
	    {
	      z[id] = PREC_DADD(PREC_DMUL(alpha, x[id]), PREC_DMUL(beta,y[id]));
	    }
	}
}

void spgpuDaxpby(spgpuHandle_t handle,
	__device double *z,
	int n,
	double beta,
	__device double *y,
	double alpha,
	__device double* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuDaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}
#else

__global__ void spgpuDaxpby_krn(double *z, int n, double beta, double *y, double alpha, double* x)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	
	if (id < n)
	{
		// Since z, x and y are accessed with the same offset by the same thread,
		// and the write to z follows the x and y read, x, y and z can share the same base address (in-place computing).

		if (beta == 0.0)
			z[id] = PREC_DMUL(alpha,x[id]);
		else
			z[id] = PREC_DADD(PREC_DMUL(alpha, x[id]), PREC_DMUL(beta,y[id]));
	}
}


void spgpuDaxpby_(spgpuHandle_t handle,
	__device double *z,
	int n,
	double beta,
	__device double *y,
	double alpha,
	__device double* x)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;

	dim3 block(BLOCK_SIZE);
	dim3 grid(msize);

	spgpuDaxpby_krn<<<grid, block, 0, handle->currentStream>>>(z, n, beta, y, alpha, x);
}

void spgpuDaxpby(spgpuHandle_t handle,
	__device double *z,
	int n,
	double beta,
	__device double *y,
	double alpha,
	__device double* x)
{
	int maxNForACall = max(handle->maxGridSizeX, BLOCK_SIZE*handle->maxGridSizeX);
	while (n > maxNForACall) //managing large vectors
	{
		spgpuDaxpby_(handle, z, maxNForACall, beta, y, alpha, x);
		
		x = x + maxNForACall;
		y = y + maxNForACall;
		z = z + maxNForACall;
		n -= maxNForACall;
	}
	
	spgpuDaxpby_(handle, z, n, beta, y, alpha, x);

	cudaCheckError("CUDA error on daxpby");
}

#endif
void spgpuDmaxpby(spgpuHandle_t handle,
		  __device double *z,
		  int n,
		  double beta,
		  __device double *y,
		  double alpha,
		  __device double* x, 
		  int count, int pitch)
{

  for (int i=0; i<count; i++)
    spgpuDaxpby(handle, z+pitch*i, n, beta, y+pitch*i, alpha, x+pitch*i);
  
}
