#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuSxyzw_krn(int n, float  a, float  b,
			       float  c, float  d,
			       float  e, float  f,
			       float * x, float  *y,
			       float  *z, float  *w)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	float  ty, tz;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  ty    = PREC_FADD(PREC_FMUL(a, x[id]), PREC_FMUL(b,y[id]));
	  tz    = PREC_FADD(PREC_FMUL(c, ty), PREC_FMUL(d,z[id]));
	  w[id] = PREC_FADD(PREC_FMUL(e, tz), PREC_FMUL(f,w[id]));
	  y[id] = ty;
	  z[id] = tz;
	}
}


void spgpuSxyzw(spgpuHandle_t handle,
		int n,
		float  a, float  b,
		float  c, float  d,
		float  e, float  f,
		__device float * x,
		__device float * y,
		__device float * z,
		__device float *w)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuSxyzw_krn<<<grid, block, 0, handle->currentStream>>>(n, a,b,c,d,e,f,
								  x, y, z,w);
}

