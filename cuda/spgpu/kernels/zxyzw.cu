#include "hip/hip_runtime.h"
/*
 * spGPU - Sparse matrices on GPU library.
 * 
 * Copyright (C) 2010 - 2012 
 *     Davide Barbieri - University of Rome Tor Vergata
 *
 * This program is free software; you can redistribute it and/or
 * modify it under the terms of the GNU General Public License
 * version 3 as published by the Free Software Foundation.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 */

#include "cudadebug.h"
#include "cudalang.h"
#include <hip/hip_runtime.h>

extern "C"
{
#include "core.h"
#include "vector.h"
  int getGPUMultiProcessors();
  int getGPUMaxThreadsPerMP();
}


#include "debug.h"

#define BLOCK_SIZE 512

__global__ void spgpuZxyzw_krn(int n, hipDoubleComplex  a, hipDoubleComplex  b,
			       hipDoubleComplex  c, hipDoubleComplex  d,
			       hipDoubleComplex  e, hipDoubleComplex  f,
			       hipDoubleComplex * x, hipDoubleComplex  *y,
			       hipDoubleComplex  *z, hipDoubleComplex  *w)
{
	int id = threadIdx.x + BLOCK_SIZE*blockIdx.x;
	unsigned int gridSize = blockDim.x * gridDim.x;
	hipDoubleComplex  ty, tz;
	for ( ; id < n; id +=gridSize)
		//if (id,n) 
	{

	  ty    = hipCfma(a, x[id], hipCmul(b,y[id]));
	  tz    = hipCfma(c, ty, hipCmul(d,z[id]));
	  w[id] = hipCfma(e, tz, hipCmul(f,w[id]));
	  y[id] = ty;
	  z[id] = tz;
	}
}


void spgpuZxyzw(spgpuHandle_t handle,
		int n,
		hipDoubleComplex  a, hipDoubleComplex  b,
		hipDoubleComplex  c, hipDoubleComplex  d,
		hipDoubleComplex  e, hipDoubleComplex  f,
		__device hipDoubleComplex * x,
		__device hipDoubleComplex * y,
		__device hipDoubleComplex * z,
		__device hipDoubleComplex *w)
{
	int msize = (n+BLOCK_SIZE-1)/BLOCK_SIZE;
	int num_mp, max_threads_mp, num_blocks_mp, num_blocks;
	dim3 block(BLOCK_SIZE);
	num_mp         = getGPUMultiProcessors();
	max_threads_mp = getGPUMaxThreadsPerMP();
	num_blocks_mp  = max_threads_mp/BLOCK_SIZE;
	num_blocks     = num_blocks_mp*num_mp;
	dim3 grid(num_blocks);

	spgpuZxyzw_krn<<<grid, block, 0, handle->currentStream>>>(n, a,b,c,d,e,f,
								  x, y, z,w);
}

